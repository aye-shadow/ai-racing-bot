#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define VERBOSE 0
#define NOEDGE 255
#define POSSIBLE_EDGE 128
#define EDGE 0

void follow_edges(unsigned char *edgemapptr, short *edgemagptr, short lowval, int cols) {
    short *tempmagptr;
    unsigned char *tempmapptr;
    int i;
    int x[8] = {1, 1, 0, -1, -1, -1, 0, 1};
    int y[8] = {0, 1, 1, 1, 0, -1, -1, -1};

    for(i = 0; i < 8; i++){
        tempmapptr = edgemapptr - y[i]*cols + x[i];
        tempmagptr = edgemagptr - y[i]*cols + x[i];

        if((*tempmapptr == POSSIBLE_EDGE) && (*tempmagptr > lowval)){
            *tempmapptr = (unsigned char) EDGE;
            follow_edges(tempmapptr, tempmagptr, lowval, cols);
        }
    }
}

void apply_hysteresis(short *mag, unsigned char *nms, int rows, int cols,
                      float tlow, float thigh, unsigned char *edge) {
    int r, c, pos, numedges, lowthreshold, highthreshold, i, hist[32768];
    short maximum_mag;
    
    for(r = 0, pos = 0; r < rows; r++){
        for(c = 0; c < cols; c++, pos++){
            if(nms[pos] == POSSIBLE_EDGE)
                edge[pos] = POSSIBLE_EDGE;
            else
                edge[pos] = NOEDGE;
        }
    }
    
    for(r = 0, pos = 0; r < rows; r++, pos += cols){
        edge[pos] = NOEDGE;
        edge[pos + cols - 1] = NOEDGE;
    }
    pos = (rows - 1) * cols;
    for(c = 0; c < cols; c++, pos++){
        edge[c] = NOEDGE;
        edge[pos] = NOEDGE;
    }
    
    for(r = 0; r < 32768; r++)
        hist[r] = 0;
    for(r = 0, pos = 0; r < rows; r++){
        for(c = 0; c < cols; c++, pos++){
            if(edge[pos] == POSSIBLE_EDGE)
                hist[mag[pos]]++;
        }
    }
    
    for(r = 1, numedges = 0; r < 32768; r++){
        if(hist[r] != 0)
            maximum_mag = r;
        numedges += hist[r];
    }
    int highcount = (int)(numedges * thigh + 0.5);
    r = 1;
    numedges = hist[1];
    while((r < (maximum_mag - 1)) && (numedges < highcount)){
        r++;
        numedges += hist[r];
    }
    highthreshold = r;
    lowthreshold = (int)(highthreshold * tlow + 0.5);
    
    if(VERBOSE){
        printf("Input low and high fractions: %f %f\n", tlow, thigh);
        printf("Computed thresholds: %d %d\n", lowthreshold, highthreshold);
    }
    
    for(r = 0, pos = 0; r < rows; r++){
        for(c = 0; c < cols; c++, pos++){
            if((edge[pos] == POSSIBLE_EDGE) && (mag[pos] >= highthreshold)){
                edge[pos] = EDGE;
                follow_edges(&edge[pos], &mag[pos], lowthreshold, cols);
            }
        }
    }
    
    for(r = 0, pos = 0; r < rows; r++){
        for(c = 0; c < cols; c++, pos++){
            if(edge[pos] != EDGE)
                edge[pos] = NOEDGE;
        }
    }
}

int read_pgm_image(char *infilename, unsigned char **image, int *rows, int *cols) {
    FILE *fp;
    char buf[71];

    if(infilename == NULL)
        fp = stdin;
    else {
        if((fp = fopen(infilename, "r")) == NULL){
            fprintf(stderr, "Error reading the file %s in read_pgm_image().\n", infilename);
            return(0);
        }
    }
    
    fgets(buf, 70, fp);
    if(strncmp(buf, "P5", 2) != 0){
        fprintf(stderr, "The file %s is not in PGM format in read_pgm_image().\n", infilename);
        if(fp != stdin)
            fclose(fp);
        return(0);
    }
    do { fgets(buf, 70, fp); } while(buf[0] == '#');
    sscanf(buf, "%d %d", cols, rows);
    do { fgets(buf, 70, fp); } while(buf[0] == '#');
    
    *image = (unsigned char *) malloc((*rows) * (*cols));
    if(*image == NULL){
        fprintf(stderr, "Memory allocation failure in read_pgm_image().\n");
        if(fp != stdin)
            fclose(fp);
        return(0);
    }
    if((*rows) != fread((*image), (*cols), (*rows), fp)){
        fprintf(stderr, "Error reading the image data in read_pgm_image().\n");
        if(fp != stdin)
            fclose(fp);
        free(*image);
        return(0);
    }
    if(fp != stdin)
        fclose(fp);
    return(1);
}

int write_pgm_image(char *outfilename, unsigned char *image, int rows, int cols, char *comment, int maxval) {
    FILE *fp;
    if(outfilename == NULL)
        fp = stdout;
    else {
        if((fp = fopen(outfilename, "w")) == NULL){
            fprintf(stderr, "Error writing the file %s in write_pgm_image().\n", outfilename);
            return(0);
        }
    }
    fprintf(fp, "P5\n%d %d\n", cols, rows);
    if(comment != NULL && strlen(comment) <= 70)
        fprintf(fp, "# %s\n", comment);
    fprintf(fp, "%d\n", maxval);
    
    if(rows != fwrite(image, cols, rows, fp)){
        fprintf(stderr, "Error writing the image data in write_pgm_image().\n");
        if(fp != stdout)
            fclose(fp);
        return(0);
    }
    if(fp != stdout)
        fclose(fp);
    return(1);
}

#define BOOSTBLURFACTOR 90.0f
#define BLOCK_SIZE 16

__global__ void gaussianHorizontal(const unsigned char* input, float* temp, int rows, int cols,
                                     const float* kernel, int kernelRadius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < cols && y < rows) {
        float sum = 0.0f, weight = 0.0f;
        for (int k = -kernelRadius; k <= kernelRadius; k++) {
            int curX = x + k;
            if(curX >= 0 && curX < cols) {
                sum += input[y * cols + curX] * kernel[kernelRadius + k];
                weight += kernel[kernelRadius + k];
            }
        }
        temp[y * cols + x] = sum / weight;
    }
}

__global__ void gaussianVertical(const float* temp, short* smoothed, int rows, int cols,
                                   const float* kernel, int kernelRadius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < cols && y < rows) {
        float sum = 0.0f, weight = 0.0f;
        for (int k = -kernelRadius; k <= kernelRadius; k++) {
            int curY = y + k;
            if(curY >= 0 && curY < rows) {
                sum += temp[curY * cols + x] * kernel[kernelRadius + k];
                weight += kernel[kernelRadius + k];
            }
        }
        smoothed[y * cols + x] = (short)(sum * BOOSTBLURFACTOR / weight + 0.5f);
    }
}

__global__ void derivativeKernel(const short* smoothed, short* deltaX, short* deltaY,
                                 int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < cols && y < rows) {
        int idx = y * cols + x;
        if(x == 0)
            deltaX[idx] = smoothed[idx + 1] - smoothed[idx];
        else if(x == cols - 1)
            deltaX[idx] = smoothed[idx] - smoothed[idx - 1];
        else
            deltaX[idx] = smoothed[y * cols + (x + 1)] - smoothed[y * cols + (x - 1)];
        if(y == 0)
            deltaY[idx] = smoothed[idx + cols] - smoothed[idx];
        else if(y == rows - 1)
            deltaY[idx] = smoothed[idx] - smoothed[idx - cols];
        else
            deltaY[idx] = smoothed[(y + 1) * cols + x] - smoothed[(y - 1) * cols + x];
    }
}

__global__ void magnitudeKernel(const short* deltaX, const short* deltaY, short* magnitude,
                                int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < cols && y < rows) {
        int idx = y * cols + x;
        int dx = deltaX[idx], dy = deltaY[idx];
        magnitude[idx] = (short)(0.5f + sqrtf((float)(dx * dx + dy * dy)));
    }
}

__global__ void nonMaxSuppressionKernel(const short* mag, const short* deltaX, const short* deltaY,
                                        unsigned char* nms, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x > 0 && y > 0 && x < cols - 1 && y < rows - 1) {
        int idx = y * cols + x;
        short current = mag[idx];
        if(current >= mag[idx - 1] && current >= mag[idx + 1] &&
           current >= mag[idx - cols] && current >= mag[idx + cols])
            nms[idx] = POSSIBLE_EDGE;
        else
            nms[idx] = NOEDGE;
    }
}

int main(int argc, char *argv[]) {
    if(argc < 5) {
       fprintf(stderr, "Usage: %s image sigma tlow thigh\n", argv[0]);
       exit(1);
    }
    char *infilename = argv[1];
    float sigma = atof(argv[2]);
    float tlow = atof(argv[3]);
    float thigh = atof(argv[4]);

    unsigned char *image;
    int rows, cols;
    if(!read_pgm_image(infilename, &image, &rows, &cols)) {
        fprintf(stderr, "Error reading image %s\n", infilename);
        exit(1);
    }

    int kernelRadius = ceil(2.5f * sigma);
    int kernelSize = 1 + 2 * kernelRadius;
    float *h_kernel = (float*)malloc(kernelSize * sizeof(float));
    float sum = 0.0f;
    for (int i = 0; i < kernelSize; i++) {
        int x = i - kernelRadius;
        h_kernel[i] = expf(-0.5f * (x * x) / (sigma * sigma)) / (sigma * sqrtf(6.2831853f));
        sum += h_kernel[i];
    }
    for (int i = 0; i < kernelSize; i++) {
        h_kernel[i] /= sum;
    }

    unsigned char *d_image;
    hipMalloc((void**)&d_image, rows * cols * sizeof(unsigned char));
    hipMemcpy(d_image, image, rows * cols * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    float *d_kernel;
    hipMalloc((void**)&d_kernel, kernelSize * sizeof(float));
    hipMemcpy(d_kernel, h_kernel, kernelSize * sizeof(float), hipMemcpyHostToDevice);

    float *d_temp;
    hipMalloc((void**)&d_temp, rows * cols * sizeof(float));
    short *d_smoothed;
    hipMalloc((void**)&d_smoothed, rows * cols * sizeof(short));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    gaussianHorizontal<<<grid, block>>>(d_image, d_temp, rows, cols, d_kernel, kernelRadius);
    hipDeviceSynchronize();
    gaussianVertical<<<grid, block>>>(d_temp, d_smoothed, rows, cols, d_kernel, kernelRadius);
    hipDeviceSynchronize();

    short *d_deltaX, *d_deltaY;
    hipMalloc((void**)&d_deltaX, rows * cols * sizeof(short));
    hipMalloc((void**)&d_deltaY, rows * cols * sizeof(short));
    derivativeKernel<<<grid, block>>>(d_smoothed, d_deltaX, d_deltaY, rows, cols);
    hipDeviceSynchronize();

    short *d_magnitude;
    hipMalloc((void**)&d_magnitude, rows * cols * sizeof(short));
    magnitudeKernel<<<grid, block>>>(d_deltaX, d_deltaY, d_magnitude, rows, cols);
    hipDeviceSynchronize();

    unsigned char *d_nms;
    hipMalloc((void**)&d_nms, rows * cols * sizeof(unsigned char));
    nonMaxSuppressionKernel<<<grid, block>>>(d_magnitude, d_deltaX, d_deltaY, d_nms, rows, cols);
    hipDeviceSynchronize();

    unsigned char *nms = (unsigned char*)malloc(rows * cols * sizeof(unsigned char));
    hipMemcpy(nms, d_nms, rows * cols * sizeof(unsigned char), hipMemcpyDeviceToHost);
    short *h_magnitude = (short*)malloc(rows * cols * sizeof(short));
    hipMemcpy(h_magnitude, d_magnitude, rows * cols * sizeof(short), hipMemcpyDeviceToHost);

    unsigned char *edge = (unsigned char*)malloc(rows * cols * sizeof(unsigned char));
    apply_hysteresis(h_magnitude, nms, rows, cols, tlow, thigh, edge);

    char outfilename[128];
    sprintf(outfilename, "%s_cuda_s_%3.2f_l_%3.2f_h_%3.2f.pgm", infilename, sigma, tlow, thigh);
    if(!write_pgm_image(outfilename, edge, rows, cols, "", 255)) {
        fprintf(stderr, "Error writing edge image %s\n", outfilename);
        exit(1);
    }

    free(image);
    free(nms);
    free(edge);
    free(h_kernel);
    free(h_magnitude);
    hipFree(d_image);
    hipFree(d_kernel);
    hipFree(d_temp);
    hipFree(d_smoothed);
    hipFree(d_deltaX);
    hipFree(d_deltaY);
    hipFree(d_magnitude);
    hipFree(d_nms);

    return 0;
}